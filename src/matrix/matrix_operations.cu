#include "hip/hip_runtime.h"
/************************************************************************
 * Copyright (C) 2011-2013:
 *
 *	Edgardo Mejia-Roa(*), Carlos Garcia, Jose Ignacio Gomez,
 *	Manuel Prieto, Francisco Tirado and Alberto Pascual-Montano(**).
 *
 *	(*)  ArTeCS Group, Complutense University of Madrid (UCM), Spain.
 *	(**) Functional Bioinformatics Group, Biocomputing Unit,
 *		National Center for Biotechnology-CSIC, Madrid, Spain.
 *
 *	E-mail for E. Mejia-Roa: <edgardomejia@fis.ucm.es>
 *	E-mail for A. Pascual-Montano: <pascual@cnb.csic.es>
 *
 *
 * This file is part of bioNMF-mGPU..
 *
 * BioNMF-mGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * BioNMF-mGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with BioNMF-mGPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 ***********************************************************************/
/**********************************************************
 * matrix_operations.cu
 *	Routines for matrix algebraic operations and data transfers.
 *	Launches kernels on the GPU.
 *
 * NOTE: The following macro constants can be defined to modify the
 *	behavior of routines, as well as some constant and data-type definitions.
 *
 *	Timing (WARNING: They PREVENT asynchronous operations):
 *		NMFGPU_PROFILING_TRANSF: Compute timing of data transfers (should be used with NMFGPU_SYNC_TRANSF).
 *		NMFGPU_PROFILING_KERNELS: Compute timing of CUDA kernels.
 *
 *	Additional information:
 *		NMFGPU_VERBOSE_2: Shows the parameters in some routine calls.
 *
 *	Debug / Testing:
 *		NMFGPU_CPU_RANDOM: Uses the CPU (host) random generator (not the CURAND library).
 *		NMFGPU_DEBUG: Shows the result of each matrix operation and data transfer.
 *		NMFGPU_DEBUG_TRANSF: Shows the result of each data transfer.
 *		NMFGPU_DEBUG_REDUCT: Shows partial results of the reduction operation.
 *		NMFGPU_SYNC_TRANSF: Performs synchronous data transfers.
 *
 **********************************************************
 *
 * NOTE: In order to improve performance:
 *
 *	- All matrices include useless data for padding. Padded dimensions
 *	  are denoted with the 'p' character, e.g., 'Mp' (i.e., M + padding)
 *	  or 'Kp' (factorization_rank + padding).
 *
 *	- Padded dimensions are a multiple of memory_alignment
 *	  (a global variable which currently is equal to warpSize or warpSize/2).
 *
 **********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <errno.h>
#include <string.h>
#include <stdint.h>	/* [u]intmax_t */

#include <hip/hip_runtime_api.h>
#include <hiprand.h>	/* Random values */

#include "matrix/matrix_io.h"
#include "GPU_kernels.cuh"
#include "GPU_setup.cuh"
#include "matrix/matrix_operations.cuh"


///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

/*
 * Partially prints device matrix content.
 * SYNCHRONOUSLY downloads a matrix from the GPU and shows its content (data, name, headers and/or labels).
 *
 * If 'transpose' is 'true', transposes matrix as follows:
 * - Matrix dimension in memory: <ncols> rows, <nrows> columns.
 * - Matrix dimension on screen: <nrows> rows, <ncols> columns.
 * - Shows <ncols> ml->headers (as column headers) and <nrows> ml->labels (as row labels).
 *
 * ncols <= pitch, unless matrix transposing is set (in that case, nrows <= padding).
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE
 */
int show_device_matrix( real const *__restrict__ dMatrix, index_t nrows, index_t ncols, index_t pitch, bool transpose,
			struct matrix_labels const *__restrict__ ml )
{

	int status = EXIT_SUCCESS;

	// Downloads the device matrix to a temporary array and shows its content.
	real *__restrict__ const buffer = (real *) malloc( nrows * pitch * sizeof(real) );
	if ( ! buffer ) {
		int const err = errno; fflush(stdout); errno = err;
		fprintf( stderr, "\n[GPU%" PRI_IDX "] Error in HOST memory allocation (malloc): %s\nError in show_device_matrix()\n",
			device_id, strerror(errno) );
		return EXIT_FAILURE;
	}

	// Synchronous data transfer.
	hipError_t const cuda_status = hipMemcpy( buffer, dMatrix, nrows * pitch * sizeof(real), hipMemcpyDeviceToHost );
	if ( cuda_status != hipSuccess ) {
		fflush(stdout);
		fprintf( stderr, "\n[GPU%" PRI_IDX "] Error downloading DEVICE matrix (nrows=%" PRI_IDX ", ncols=%" PRI_IDX ", pitch=%"
				PRI_IDX ", transpose=%i): %s\n", device_id, nrows, ncols, pitch, transpose, hipGetErrorString(cuda_status) );
		free( buffer );
		return EXIT_FAILURE;
	}
	/* Same code using CUBLAS:
	 *	hipblasStatus_t cublas_status = hipblasGetVector( nrows * pitch, sizeof(real), dMatrix, 1, buffer, 1 );
	 */

	if ( transpose )
		status = matrix_show( buffer, ncols, nrows, pitch, true, ml );
	else
		status = matrix_show( buffer, nrows, ncols, pitch, false, ml );

	free( buffer );

	return status;

} // show_device_matrix

/////////////////////////////////////////////////////////////////////

/*
 * Partially prints device matrix content (INTEGER version).
 * SYNCHRONOUSLY downloads a matrix from the GPU and shows its content (data, name, headers and/or labels).
 *
 * If 'transpose' is 'true', transposes matrix as follows:
 * - Matrix dimension in memory: <ncols> rows, <nrows> columns.
 * - Matrix dimension on screen: <nrows> rows, <ncols> columns.
 * - Shows <ncols> ml->headers (as column headers) and <nrows> ml->labels (as row labels).
 *
 * ncols <= pitch, unless matrix transposing is set (in that case, nrows <= padding).
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE
 */
int show_device_matrix_int( index_t const *__restrict__ dMatrix, index_t nrows, index_t ncols, index_t pitch, bool transpose,
			struct matrix_labels const *__restrict__ ml )
{

	int status = EXIT_SUCCESS;

	// Downloads the device matrix to a temporary array and shows its content.
	index_t *__restrict__ const buffer = (index_t *) malloc( nrows * pitch * sizeof(index_t) );
	if ( ! buffer ) {
		int const err = errno; fflush(stdout); errno = err;
		fprintf( stderr, "\n[GPU%" PRI_IDX "] Error in HOST memory allocation (malloc): %s\nError in show_device_matrix_int()\n",
			device_id, strerror(errno) );
		return EXIT_FAILURE;
	}

	// *SYNCHRONOUS* data transfer.
	hipError_t const cuda_status = hipMemcpy( buffer, dMatrix, nrows * pitch * sizeof(index_t), hipMemcpyDeviceToHost );
	if ( cuda_status != hipSuccess ) {
		fflush(stdout);
		fprintf( stderr, "\n[GPU%" PRI_IDX "] Error downloading DEVICE matrix of INTEGERS (nrows=%" PRI_IDX ", ncols=%" PRI_IDX
				", pitch=%" PRI_IDX ", transpose=%i): %s\n", device_id, nrows, ncols, pitch, transpose,
				hipGetErrorString(cuda_status) );
		free( buffer );
		return EXIT_FAILURE;
	}
	/* Same code using CUBLAS:
	 *	hipblasStatus_t cublas_status = hipblasGetVector( nrows * pitch, sizeof(index_t), dMatrix, 1, buffer, 1 );
	 */

	if ( transpose )
		status = matrix_int_show( buffer, ncols, nrows, pitch, true, ml );
	else
		status = matrix_int_show( buffer, nrows, ncols, pitch, false, ml );

	free( buffer );

	return status;

} // show_device_matrix_int

///////////////////////////////////////////////////////////////////////////////

/*
 * d_A = random_value
 *
 * width <= padding
 *
 * If NMFGPU_DEBUG || NMFGPU_VERBOSE_2:
 *	transpose: 'True' if matrix is matrix is transposed.
 *
 * If 'event_A' is non-NULL, the operation is recorded as an event.
 *
 * WARNING: Requires the CURAND Library properly initialized.
 */
void matrix_random( real *__restrict__ d_A, index_t height, index_t width, index_t padding,
			#if NMFGPU_DEBUG || NMFGPU_VERBOSE_2
				bool transpose, char const *__restrict__ const matrix_name,
			#endif
			hipStream_t stream_A, hipEvent_t *__restrict__ event_A )
{

	#if ! NMFGPU_CPU_RANDOM

		#if NMFGPU_VERBOSE_2
			if (! device_id)
				printf("\nSetting random values to matrix '%s' (height=%" PRI_IDX ", width=%" PRI_IDX ", padding=%" PRI_IDX
					", transpose=%i)\n", matrix_name, height, width, padding, transpose );
		#endif

		#if NMFGPU_DEBUG
			hiprandStatus_t curand_status = HIPRAND_STATUS_SUCCESS;
		#endif

		// ----------------------------------

		// Sets the stream

		#if NMFGPU_DEBUG
			curand_status =
		#endif

			hiprandSetStream( curand_generator, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( curand_status != HIPRAND_STATUS_SUCCESS ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error setting stream for CURAND kernel launches (matrix %s): ",
					device_id, matrix_name );
				printCurandErrorString( curand_status );
			}
		#endif
		///////////////////////////////

		// ----------------------------------

		// Generates random values.

		size_t const size = height * padding;

		#if NMFGPU_DEBUG
			curand_status =
		#endif

			CURAND_GENERATE_UNIFORM_REAL( curand_generator, d_A, size );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( curand_status != HIPRAND_STATUS_SUCCESS ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error generating random values for matrix %s: ", device_id, matrix_name );
				printCurandErrorString( curand_status );
			}
			printf( "\n--- [GPU%" PRI_IDX "] Random values on matrix %s (height=%" PRI_IDX ", width=%" PRI_IDX ", padding=%"
				PRI_IDX ", transpose=%i): ---\n", device_id, matrix_name, height, width, padding, transpose );
			check_cuda_status();
			show_device_matrix( d_A, height, width, padding, transpose, NULL );
		#endif
		/////////////////////////////

		// ----------------------------------

		// Records the previous operation on stream_A as 'event_A'
		if ( event_A ) {
			#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
				hipError_t cuda_status =
			#endif

				hipEventRecord( *event_A, stream_A );

				///////////////////////////////
				#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
					if ( cuda_status != hipSuccess ) {
						fflush(stdout);
						fprintf(stderr, "\n[GPU%" PRI_IDX "] Error recording CUDA event: %s\nError in "
							"matrix_random(%s,height=%" PRI_IDX ", width=%" PRI_IDX ", padding=%" PRI_IDX
							", transpose=%i).\n", device_id, hipGetErrorString(cuda_status),height, width,
							padding, transpose );
					}
				#endif
				///////////////////////////////
		}

		// ----------------------------------

		#if NMFGPU_VERBOSE_2
		if (! device_id)
			printf("\nSetting random values to matrix '%s' (height=%" PRI_IDX ", width=%" PRI_IDX ", padding=%" PRI_IDX
				", transpose=%i)... Done.\n", matrix_name, height, width, padding, transpose );
		#endif

	#endif /* NMFGPU_CPU_RANDOM */

} //matrix_random

///////////////////////////////////////////////////////////////////////////////

/*
 * d_accum_A[ i ] = SUM( d_A[ i ][...] )
 *
 * Reduces matrix d_A to a row.
 *
 * d_Tmp: Temporary storage. Ignored if height <= 2
 * size_of( d_Tmp ) <= (height/REDUCE_TO_ROW__ITEMS_PER_THREAD) * pitch
 * length( d_accum_A ) >= pitch
 *
 * 'pitch' must be a multiple of 'memory_alignment', and <= maxThreadsPerBlock.
 *
 * The operation is recorded with "event_reduction".
 *
 * WARNING:
 *	- On Compute Capability 1.x:
 *		height < prev_power_2(maxBlockHeight_pitch) * REDUCE_TO_ROW__ITEMS_PER_THREAD * (2**24)
 *		('REDUCE_TO_ROW__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
 */
void matrix_to_row( real const *__restrict__ d_A, index_t height, index_t pitch,
		#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
		index_t width, char const *__restrict__ const matrix_name,
		#endif
		real *__restrict__ d_Tmp, real *__restrict__ d_accum_A, hipStream_t stream_AccA )
{

	///////////////////////////////
	#if NMFGPU_DEBUG_REDUCT
		if ( ! device_id )
			printf("\n--- Begin of matrix_to_row(computeCapability=%" PRI_IDX ", width=%" PRI_IDX ", pitch=%" PRI_IDX
				", height=%" PRI_IDX ") on %s: ---\n", computeCapability, width, pitch, height, matrix_name);
	#endif
	///////////////////////////////

	// ----------------------------------------

	// Event for this operation.
	hipEvent_t event_AccA = event_reduction;

	// ----------------------------------------

	if ( height > 1 ) {

		#if NMFGPU_PROFILING_KERNELS
			index_t timing_index = 0;	// Non-extended grid (0), extended grid (1), single block (2), copy (3)
		#endif

		index_t const matrix_size = pitch * height;

		/* Uses a block width equal to <pitch>.
		 *
		 * Each block reduces, at least, <REDUCE_TO_ROW__ITEMS_PER_THREAD> times its height
		 * ('REDUCE_TO_ROW__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
		 *
		 *	Please, see the reduction example in CUDA SDK for details.
		 */

		index_t block_height = prev_power_2( maxBlockHeight_pitch );		// A power of two.

		index_t const abh = block_height * REDUCE_TO_ROW__ITEMS_PER_THREAD;	// "Active" block height.


		// Grid dimensions
		index_t grid_length = 1;	// <= maxGridSizeX
		index_t grid_extension = 1;	// <= maxGridSizeY, and <= grid_length

		/* "grid_extension" is required only if matrix_size > (maxGridSizeX * abh * pitch).
		 *
		 * On Compute Capability >= 3.0:
		 *	It never happens if (IDX_MAX / maxGridSizeX) < memory_alignment
		 *
		 * On Compute Capability 1.x:
		 *	"Grid extension" is also required if matrix_size >= 2**24
		 */
		uintmax_t max_size = maxGridSizeX * abh * pitch;	// It may be > IDX_MAX on Compute Capability >= 3.0
		if ( ( computeCapability == 1 ) * ( max_size >= (1 << 24) ) )
			max_size = (1 << 24) - 1;


		// No "grid extension" required
		if ( (uintmax_t) matrix_size <= max_size ) {

			grid_length = (height + abh - 1) / abh;

			#if NMFGPU_PROFILING_KERNELS
				timing_index = 0;	// Non-extended grid.
			#endif

			/* If there is not enough work for more than two blocks, uses just a single one.
			 * Such block will iteratively read data from global memory.
			 */
			if ( grid_length <= 2 ) {

				// Tries to use a block height as large as possible.

				index_t const max_block_height1 = height / REDUCE_TO_ROW__ITEMS_PER_THREAD;

				index_t const max_block_height2 = maxThreadsPerBlock / pitch;

				block_height = prev_power_2( MIN( max_block_height1, max_block_height2 ) );	// A power of 2

				grid_length = 1;

				#if NMFGPU_PROFILING_KERNELS
					timing_index = 2;	// Single-block mode.
				#endif

			}

		// "Extended" grid.
		} else {

			/* Grid geometry:
			 *
			 * (grid_extension * grid_length * abh) >= height.
			 *
			 * That is,
			 *	grid_extension * grid_length >= ceil( height / abh ).
			 *
			 * So, we can use:
			 *	grid_extension == grid_length == ceil( sqrt( ceil(height/abh) ) ),
			 *
			 * but this may lead to an excessive number of idle thread blocks. That is,
			 *	(grid_extension * grid_length * abh) >> height.
			 *
			 * Instead,
			 *	grid_extension	= ceil( height / (maxGridSizeX	 * abh) ), which is << maxGridSizeY
			 *	grid_length	= ceil( height / (grid_extension * abh) ), which is <= maxGridSizeX
			 *
			 * is more accurate.
			 *
			 * Note that grid_extension <= grid_length
			 */

			// Grid "extension"
			index_t const gh = maxGridSizeX * abh;
			grid_extension = ( height + gh - 1 ) / gh;	// << maxGridSizeY

			// Grid "length"
			index_t const gw = grid_extension * abh;
			grid_length = ( height + gw - 1 ) / gw;		// <= maxGridSizeX

			#if NMFGPU_PROFILING_KERNELS
				timing_index = 1;	// Extended grid
			#endif

		} // If grid extension is required

		// ---------------------------

		#if NMFGPU_PROFILING_KERNELS
			start_cuda_timer();
		#endif

			/* d_A[ height ][ pitch ] is reduced using a grid of (grid_extension * grid_length) blocks.
			 *
			 * d_Tmp[ grid_extension*grid_length ][ pitch ] is used as a temporary storage.
			 */
			reduce_to_row( d_A, pitch, d_Tmp, block_height, grid_extension, grid_length, matrix_size, stream_AccA, d_accum_A );

			///////////////////////////////
			#if NMFGPU_DEBUG_REDUCT
				if ( ! device_id )
					check_cuda_status();
			#endif
			///////////////////////////////

		#if NMFGPU_PROFILING_KERNELS
			stop_cuda_timer_cnt( &reduce_timing[ timing_index ], matrix_size, 1 );
		#endif

		// ---------------------------

		/* On Compute Capability < 1.2, a second call is required to finish the sum in
		 * d_Tmp[ grid_extension*grid_length ][ pitch ].
		 * Such call is performed in "single-block" mode.
		 */
		if ( (computeCapability == 1) * (computeCapability_minor < 2) * (grid_length > 1) ) {

			///////////////////////////////
			#if NMFGPU_DEBUG_REDUCT
				// Resulting d_Tmp from previous stage:
				if ( ! device_id ) {
					printf("\n---Resulting d_Tmp (height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
						",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX "):---\n",
						height, width, pitch, block_height, grid_extension, grid_length );
					check_cuda_status();
					show_device_matrix( d_Tmp, (grid_extension * grid_length), width, pitch, false, NULL );
				}
			#endif
			///////////////////////////////

			// ---------------------------

			#if NMFGPU_PROFILING_KERNELS
				start_cuda_timer();
			#endif

				/* d_Tmp[ grid_extension*grid_length ][ pitch ] is reduced with a single block.
				 * No temporary storage is required.
				 */
				index_t const Tmp_size = grid_extension * grid_length;
				reduce_to_row( d_Tmp, pitch, NULL, block_height, 1, 1, Tmp_size, stream_AccA, d_accum_A );

				///////////////////////////////
				#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
					if ( ! device_id )
						check_cuda_status();
				#endif
				///////////////////////////////

			#if NMFGPU_PROFILING_KERNELS
				stop_cuda_timer_cnt( &reduce_timing[2], matrix_size, 1 );
			#endif

		} // If a second call is required.

	} else { // (height == 1)

		/* Just copies d_A[] to d_accum_A[]. */

		#if NMFGPU_PROFILING_KERNELS
			start_cuda_timer();
		#endif

			#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
				hipError_t cuda_status =
			#endif

				hipMemcpyAsync( d_accum_A, d_A, pitch * sizeof(real), hipMemcpyDeviceToDevice, stream_AccA );

			/* Same code using CUBLAS:
			 *	cublas_status = hipblasSetStream( cublas_handle, stream_AccA );
			 *	cublas_status = CUBLAS_R_COPY( cublas_handle, pitch, d_A, 1, d_accum_A, 1 );
			 */

			///////////////////////////////
			#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
				check_cuda_status_st( cuda_status );
			#endif
			///////////////////////////////

		#if NMFGPU_PROFILING_KERNELS
			stop_cuda_timer_cnt( &reduce_timing[3], pitch, 1 );
		#endif

	} // if ( height > 1 )


	///////////////////////////////
	#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
		printf("\n--- [GPU%" PRI_IDX "] Resulting accumulator (length=%" PRI_IDX ",pitch=%" PRI_IDX ") for matrix %s: ---\n",
			device_id, width, pitch, matrix_name );
		check_cuda_status();
		show_device_matrix( d_accum_A, 1, width, pitch, false, NULL );
	#endif
	///////////////////////////////

	// ------------------------------------

	// Records the previous operation on stream_AccA as 'event_AccA'
	{
		#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipEventRecord( event_AccA, stream_AccA );

			///////////////////////////////
			#if NMFGPU_DEBUG_REDUCT || NMFGPU_DEBUG
				if ( cuda_status != hipSuccess ) {
					fflush(stdout);
					fprintf(stderr, "\n[GPU%" PRI_IDX "] Error recording CUDA event: %s\nError in matrix2row(%s, width=%"
						PRI_IDX ", pitch=%" PRI_IDX ").\n", device_id, hipGetErrorString(cuda_status),
						matrix_name, width, pitch );
				}
			#endif
			///////////////////////////////
	}

} // matrix_to_row

///////////////////////////////////////////////////////////////////////////////

/*
 * d_A = d_B <op> d_A
 *
 * <op> is "./" or "-"
 *
 * div_operand: 'True' if operation to perform is a floating-point division.
 *		Otherwise, a subtraction is performed.
 *
 * Kernel launch is delayed upon event "event_B" completes.
 * Then, the operation is registered using the same event object.
 *
 * 'pitch' must be a multiple of 'memory_alignment'.
 *
 * WARNING:
 *	- On Compute Capability 1.x:
 *		matrix_size < threadsPerBlock * DIV_SUB__ITEMS_PER_THREAD * (2**24)
 *		('DIV_SUB__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h")
 */
void matrix_div_sub( real *__restrict__ d_A, real const *__restrict__ d_B, index_t height, index_t pitch,
			#if NMFGPU_DEBUG
				index_t width, char const *__restrict__ const matrix_name_A,
				char const *__restrict__ const matrix_name_B,
			#endif
			bool div_operand,
			#if NMFGPU_PROFILING_KERNELS
				timing_data_t *__restrict__ td,
			#endif
			hipStream_t stream_A, hipEvent_t event_B )
{

	#if NMFGPU_PROFILING_KERNELS
		index_t timing_index = 0;
	#endif

	index_t const matrix_size = pitch * height;

	// ------------------------------------------

	/* Uses 1-D blocks.
	 *
	 * Each block processes up to <DIV_SUB__ITEMS_PER_THREAD> times its size
	 * ('DIV_SUB__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
	 */

	index_t const block_size = threadsPerBlock;

	index_t const act_bs = block_size * DIV_SUB__ITEMS_PER_THREAD;		// "Active" block size


	// Grid dimensions
	index_t grid_length = 1;	// <= maxGridSizeX
	index_t grid_extension = 1;	// <= maxGridSizeY, and <= grid_length

	/* "grid_extension" is required only if matrix_size > (maxGridSizeX * act_bs).
	 *
	 * On Compute Capability >= 3.0:
	 *	It never happens if (IDX_MAX / maxGridSizeX) < memory_alignment
	 *
	 * On Compute Capability 1.x:
	 *	"Grid extension" is also required if matrix_size >= 2**24
	 */
	uintmax_t max_size = maxGridSizeX * act_bs;	// It may be > IDX_MAX on Compute Capability >= 3.0
	if ( ( computeCapability == 1 ) * ( max_size >= (1 << 24) ) )
		max_size = (1 << 24) - 1;


	// No "grid extension" required
	if ( (uintmax_t) matrix_size <= max_size ) {

		grid_length = (matrix_size + act_bs - 1) / act_bs;

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 0;
		#endif

	// "Extended" grid.
	} else {

		/* Grid geometry:
		 *
		 * (grid_extension * grid_length * act_bs) >= matrix_size.
		 *
		 * That is,
		 *	grid_extension * grid_length >= ceil( matrix_size / act_bs ).
		 *
		 * So, we can use:
		 *	grid_extension == grid_length == ceil( sqrt( ceil(matrix_size/act_bs) ) ),
		 *
		 * but this may lead to an excessive number of idle thread blocks. That is,
		 *	(grid_extension * grid_length * act_bs) >> matrix_size.
		 *
		 * Instead,
		 *	grid_extension	= ceil( matrix_size / (maxGridSizeX   * act_bs) ), which is << maxGridSizeY
		 *	grid_length	= ceil( matrix_size / (grid_extension * act_bs) ), which is <= maxGridSizeX
		 *
		 * is more accurate.
		 *
		 * Note that grid_extension <= grid_length
		 */

		// Grid "extension"
		index_t const gh = maxGridSizeX * act_bs;
		grid_extension = ( height + gh - 1 ) / gh;	// << maxGridSizeY

		// Grid "length"
		index_t const gw = grid_extension * act_bs;
		grid_length = ( height + gw - 1 ) / gw;		// <= maxGridSizeX

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 1;
		#endif

	} // If grid extension is required

	// ------------------------------------------

	// Delays kernel launch until d_B[] is ready.

	{

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipStreamWaitEvent( stream_A, event_B, 0 );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] hipStreamWaitEvent: %s\nError in matrix_div_sub(%s %s %s, height=%" PRI_IDX
					", pitch=%" PRI_IDX ").\n", device_id, hipGetErrorString(cuda_status), matrix_name_A,
					( div_operand ? "./" : "-" ), matrix_name_B, height, pitch );
			}
		#endif
		///////////////////////////////
	}

	// ------------------------------------------

	// Launches the kernel.

	#if NMFGPU_PROFILING_KERNELS
		start_cuda_timer();
	#endif

		div_sub( d_A, d_B, matrix_size, block_size, grid_extension, grid_length, div_operand, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG
			printf("\n--- [GPU%" PRI_IDX "] Resulting %s = %s %s %s (height=%" PRI_IDX ", width=%" PRI_IDX ", pitch=%" PRI_IDX
				",block_size=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX "): ---\n", device_id,
				matrix_name_A, matrix_name_B, ( div_operand ? "./" : "-" ), matrix_name_A, height, width, pitch, block_size,
				grid_extension, grid_length );
			check_cuda_status();
			show_device_matrix( d_A, height, width, pitch, false, NULL );
		#endif
		///////////////////////////////

	#if NMFGPU_PROFILING_KERNELS
		stop_cuda_timer_cnt( &td[ timing_index ], matrix_size, 1 );
	#endif

	// ------------------------------------------

	// Records the operations as an event (reuses the event object).
	{

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipEventRecord( event_B, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error recording CUDA event: %s\nError in matrix_div(%s %s %s, height=%"
						PRI_IDX ", pitch=%" PRI_IDX ").\n", device_id, hipGetErrorString(cuda_status), matrix_name_A,
						( div_operand ? "./" : "-" ), matrix_name_B, height, pitch );
			}
		#endif
		///////////////////////////////

	}

} // matrix_div_sub

/////////////////////////////////////////////////////////////////////

/*
 * d_A[i][j] = d_A[i][j] .* d_Aux[i][j] ./ d_accum_B[j]
 *
 * length(d_accum_B) >= pitch
 *
 * Kernel launch is delayed upon event "event_accB" completes.
 * Then, the operation is registered using the same event object.
 *
 * 'pitch' must be a multiple of 'memory_alignment', and <= maxThreadsPerBlock.
 *
 * WARNING:
 *	- On Compute Capability 1.x:
 *		height < maxBlockHeight_pitch * MUL_DIV__ITEMS_PER_THREAD * (2**24)
 *		('MUL_DIV__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
 */
void matrix_mul_div( real *__restrict__ d_A, real const *__restrict__ d_Aux, real const *__restrict__ d_accum_B, index_t height, index_t pitch,
			#if NMFGPU_DEBUG
				index_t width, bool transpose, char const *__restrict__ const matrix_name_A,
				char const *__restrict__ const matrix_name_Aux, char const *__restrict__ const matrix_name_accB,
			#endif
			hipStream_t stream_A )
{

	/* CPU code:
	 * for ( index_t i=0 ; i<height ; i++ )
	 *	for ( index_t j=0 ; j<pitch ; j++ )
	 *		d_A[i][j] = d_A[i][j] * d_Aux[i][j] / d_accum_B[j];
	 */

	// ------------------------------------------

	#if NMFGPU_PROFILING_KERNELS
		index_t timing_index = 0;
	#endif

	index_t const matrix_size = pitch * height;

	// ------------------------------------------

	/* Uses a block width equal to <pitch>.
	 *
	 * Each block processes up to <MUL_DIV__ITEMS_PER_THREAD> times its height
	 * ('MUL_DIV__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
	 */

	index_t const abh = maxBlockHeight_pitch * REDUCE_TO_ROW__ITEMS_PER_THREAD;	// "Active" block height.


	// Grid dimensions
	index_t grid_length = 1;	// <= maxGridSizeX
	index_t grid_extension = 1;	// <= maxGridSizeY, and <= grid_length

	/* "grid_extension" is required only if matrix_size > (maxGridSizeX * abh * pitch).
	 *
	 * On Compute Capability >= 3.0:
	 *	It never happens if (IDX_MAX / maxGridSizeX) < memory_alignment
	 *
	 * On Compute Capability 1.x:
	 *	"Grid extension" is also required if matrix_size >= 2**24
	 */
	uintmax_t max_size = maxGridSizeX * abh * pitch;	// It may be > IDX_MAX on Compute Capability >= 3.0
	if ( ( computeCapability == 1 ) * ( max_size >= (1 << 24) ) )
		max_size = (1 << 24) - 1;


	// No "grid extension" required
	if ( (uintmax_t) matrix_size <= max_size ) {

		grid_length = (height + abh - 1) / abh;

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 0;
		#endif

	// "Extended" grid.
	} else {

		/* Grid geometry:
		 *
		 * (grid_extension * grid_length * abh) >= height.
		 *
		 * That is,
		 *	grid_extension * grid_length >= ceil( height / abh ).
		 *
		 * So, we can use:
		 *	grid_extension == grid_length == ceil( sqrt( ceil(height/abh) ) ),
		 *
		 * but this may lead to an excessive number of idle thread blocks. That is,
		 *	(grid_extension * grid_length * abh) >> height.
		 *
		 * Instead,
		 *	grid_extension	= ceil( height / (maxGridSizeX	 * abh) ), which is << maxGridSizeY
		 *	grid_length	= ceil( height / (grid_extension * abh) ), which is <= maxGridSizeX
		 *
		 * is more accurate.
		 *
		 * Note that grid_extension <= grid_length
		 */

		// Grid "extension"
		index_t const gh = maxGridSizeX * abh;
		grid_extension = ( height + gh - 1 ) / gh;	// << maxGridSizeY

		// Grid "length"
		index_t const gw = grid_extension * abh;
		grid_length = ( height + gw - 1 ) / gw;		// <= maxGridSizeX

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 1;
		#endif

	} // If grid extension is required

	// ------------------------------------------

	// Delays kernel launch until d_accum_B[] is ready.
	{

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipStreamWaitEvent( stream_A, event_reduction, 0 );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error: could not delay operations until %s is ready: %s\n"
					"Error in matrix_mul_div(%s, %s, %s, height=%" PRI_IDX ",width=%" PRI_IDX ", pitch=%" PRI_IDX
					",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX ", transpose=%i\n",
					device_id, matrix_name_accB, hipGetErrorString(cuda_status), matrix_name_A, matrix_name_Aux,
					matrix_name_accB, height, width, pitch, maxBlockHeight_pitch, grid_extension, grid_length, transpose );
			}
		#endif
		///////////////////////////////
	}

	// ------------------------------------------

	#if NMFGPU_PROFILING_KERNELS
		start_cuda_timer();
	#endif

		mul_div( d_A, d_Aux, d_accum_B, pitch, matrix_size, maxBlockHeight_pitch, grid_extension, grid_length, stream_A );

			///////////////////////////////
			#if NMFGPU_DEBUG
				printf("\n--- [GPU%" PRI_IDX "] Resulting %s = %s .* %s ./ %s (height=%" PRI_IDX ",width=%" PRI_IDX
					", pitch=%" PRI_IDX ",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX
					", transpose=%i ): ---\n", device_id, matrix_name_A, matrix_name_A, matrix_name_Aux, matrix_name_accB,
					height, width, pitch, maxBlockHeight_pitch, grid_extension, grid_length, transpose );
				check_cuda_status();
				show_device_matrix( d_A, height, width, pitch, transpose, NULL );
			#endif
			///////////////////////////////

	#if NMFGPU_PROFILING_KERNELS
		stop_cuda_timer_cnt( &mul_div_timing[ timing_index ], matrix_size, 1 );
	#endif

} // matrix_mul_div

/////////////////////////////////////////////////////////////////////

/*
 * d_A = MAX( d_A , R_MIN )
 *
 * 'pitch' must be a multiple of 'memory_alignment', and <= maxThreadsPerBlock.
 *
 * If 'event_A' is non-NULL, delays the kernel launch until such event completes.
 * Then, the operation is recorded using the same event object.
 *
 * WARNING:
 *	- On Compute Capability 1.x:
 *		height < maxBlockHeight_pitch * ADJUST__ITEMS_PER_THREAD * (2**24)
 *		('ADJUST__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h")
 */
void matrix_adjust( real *__restrict__ d_A, index_t height, index_t pitch,
			#if NMFGPU_DEBUG
				index_t width, bool transpose, char const *__restrict__ const matrix_name_A,
			#endif
			hipStream_t stream_A, hipEvent_t *__restrict__ event_A )
{

	#if NMFGPU_PROFILING_KERNELS
		index_t timing_index = 0;
	#endif

	index_t const matrix_size = pitch * height;

	// ------------------------------------------

	/* Uses a block width equal to <pitch>.
	 *
	 * Each block processes up to <ADJUST__ITEMS_PER_THREAD> times its height
	 * ('ADJUST__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
	 */

	index_t const abh = maxBlockHeight_pitch * ADJUST__ITEMS_PER_THREAD;	// "Active" block height.


	// Grid dimensions
	index_t grid_length = 1;	// <= maxGridSizeX
	index_t grid_extension = 1;	// <= maxGridSizeY, and <= grid_length

	/* "grid_extension" is required only if matrix_size > (maxGridSizeX * abh * pitch).
	 *
	 * On Compute Capability >= 3.0:
	 *	It never happens if (IDX_MAX / maxGridSizeX) < memory_alignment
	 *
	 * On Compute Capability 1.x:
	 *	"Grid extension" is also required if matrix_size >= 2**24
	 */
	uintmax_t max_size = maxGridSizeX * abh * pitch;	// It may be > IDX_MAX on Compute Capability >= 3.0
	if ( ( computeCapability == 1 ) * ( max_size >= (1 << 24) ) )
		max_size = (1 << 24) - 1;


	// No "grid extension" required
	if ( (uintmax_t) matrix_size <= max_size ) {

		grid_length = (height + abh - 1) / abh;

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 0;
		#endif

	// "Extended" grid.
	} else {

		/* Grid geometry:
		 *
		 * (grid_extension * grid_length * abh) >= height.
		 *
		 * That is,
		 *	grid_extension * grid_length >= ceil( height / abh ).
		 *
		 * So, we can use:
		 *	grid_extension == grid_length == ceil( sqrt( ceil(height/abh) ) ),
		 *
		 * but this may lead to an excessive number of idle thread blocks. That is,
		 *	(grid_extension * grid_length * abh) >> height.
		 *
		 * Instead,
		 *	grid_extension	= ceil( height / (maxGridSizeX	 * abh) ), which is << maxGridSizeY
		 *	grid_length	= ceil( height / (grid_extension * abh) ), which is <= maxGridSizeX
		 *
		 * is more accurate.
		 *
		 * Note that grid_extension <= grid_length
		 */

		// Grid "extension"
		index_t const gh = maxGridSizeX * abh;
		grid_extension = ( height + gh - 1 ) / gh;	// << maxGridSizeY

		// Grid "length"
		index_t const gw = grid_extension * abh;
		grid_length = ( height + gw - 1 ) / gw;		// <= maxGridSizeX

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 1;
		#endif

	} // If grid extension is required

	// ---------------------------

	// Delays kernel launch until d_A[] is ready.
	if ( event_A ) {

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipStreamWaitEvent( stream_A, *event_A, 0 );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error: could not delay operations until %s is ready: %s\n"
					"Error in matrix_adjust(height=%" PRI_IDX ",width=%" PRI_IDX ", pitch=%" PRI_IDX
					",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX
					", transpose=%i ): ---\n", device_id, matrix_name_A, hipGetErrorString(cuda_status),
					height, width, pitch, maxBlockHeight_pitch, grid_extension, grid_length , transpose );
			}
		#endif
		///////////////////////////////
	}

	// ---------------------------

	#if NMFGPU_PROFILING_KERNELS
		start_cuda_timer();
	#endif

		adjust( d_A, pitch, matrix_size, maxBlockHeight_pitch, grid_extension, grid_length, stream_A );

			///////////////////////////////
			#if NMFGPU_DEBUG
				printf("\n--- [GPU%" PRI_IDX "] Resulting %s = MAX( %s, R_MIN ), (height=%" PRI_IDX ",width=%" PRI_IDX
					", pitch=%" PRI_IDX ",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX
					", transpose=%i ): ---\n", device_id, matrix_name_A, matrix_name_A, height, width, pitch,
					maxBlockHeight_pitch, grid_extension, grid_length , transpose );
				check_cuda_status();
				show_device_matrix( d_A, height, width, pitch, transpose, NULL );
			#endif
			///////////////////////////////

	#if NMFGPU_PROFILING_KERNELS
		stop_cuda_timer_cnt( &adjust_timing[ timing_index ], matrix_size, 1 );
	#endif

	// ---------------------------

	// Delays kernel launch until d_A[] is ready.
	if ( event_A ) {

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipEventRecord( *event_A, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr,"\n[GPU%" PRI_IDX "] Error recording CUDA event: %s\n"
					"Error in matrix_adjust(%s, height=%" PRI_IDX ",width=%" PRI_IDX ", pitch=%" PRI_IDX
					",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX ",grid_length=%" PRI_IDX
					", transpose=%i ): ---\n", device_id, hipGetErrorString(cuda_status), matrix_name_A,
					height, width, pitch, maxBlockHeight_pitch, grid_extension, grid_length , transpose );
			}
		#endif
		///////////////////////////////
	}

} // matrix_adjust

/////////////////////////////////////////////////////////////////////

/*
 * Computes the maximum value of each row in d_A[] and stores its column index in d_Idx[].
 * That is, returns d_Idx[i], such that:
 *	d_A[i][ d_Idx[i] ] == max( d_A[i][...] ).
 *
 * size_of( d_Idx ) >= height
 *
 * 'pitch' must be a multiple of 'memory_alignment', and <= maxThreadsPerBlock.
 *
 * WARNING:
 *	- On Compute Capability 1.x:
 *		height < (threadsPerBlock/block_width) * (2**24)
 */
void matrix_idx_max( real const *__restrict__ d_A, index_t width, index_t pitch, index_t height,
			#if NMFGPU_DEBUG
				bool transpose, char const *__restrict__ const matrix_name_A,
				char const *__restrict__ const matrix_name_Idx,
			#endif
			hipStream_t stream_A, index_t *__restrict__ d_Idx )
{

	#if NMFGPU_PROFILING_KERNELS
		index_t timing_index = 0;
	#endif

	index_t const matrix_size = pitch * height;

	// ------------------------------------------

	/* Each thread processes up to <IDX_MAX__ITEMS_PER_THREAD> from the same row.
	 * ('IDX_MAX__ITEMS_PER_THREAD' is a constant defined in "GPU_kernels.h").
	 *
	 * (block_width * IDX_MAX__ITEMS_PER_THREAD) must NOT be < width.
	 * Therefore, block_width >= (width / IDX_MAX__ITEMS_PER_THREAD).
	 * In addition, it must be a power of 2 <= width <= pitch.
	 */

	// block_width = Next_power_2( width / IDX_MAX__ITEMS_PER_THREAD ) >= memory_alignment

	index_t block_width = memory_alignment;
	index_t block_height = maxBlockHeight_pitch;	// If width <= memory_alignment, then pitch == memory_alignment

	if ( width > memory_alignment ) {

		block_width = ( width / IDX_MAX__ITEMS_PER_THREAD );
		block_width = prev_power_2( (block_width << 1) );	// prev_power_2( x*2 ) == next_power_2( x )
		block_width = MIN( block_width, memory_alignment );	// Note that memory_alignment is also a power of 2.

		block_height = threadsPerBlock_pitch / block_width;
	}


	// Grid dimensions
	index_t grid_length = 1;	// <= maxGridSizeX
	index_t grid_extension = 1;	// <= maxGridSizeY, and <= grid_length


	/* "grid_extension" is required only if matrix_size > (maxGridSizeX * abh * pitch).
	 *
	 * On Compute Capability >= 3.0:
	 *	It never happens if (IDX_MAX / maxGridSizeX) < memory_alignment
	 *
	 * On Compute Capability 1.x:
	 *	"Grid extension" is also required if matrix_size >= 2**24
	 */
	uintmax_t max_size = maxGridSizeX * block_height * pitch;	// It may be > IDX_MAX on Compute Capability >= 3.0
	if ( ( computeCapability == 1 ) * ( max_size >= (1 << 24) ) )
		max_size = (1 << 24) - 1;


	// No "grid extension" required
	if ( (uintmax_t) matrix_size <= max_size ) {

		grid_length = (height + block_height - 1) / block_height;

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 0;
		#endif

	// "Extended" grid.
	} else {

		/* Grid geometry:
		 *
		 * (grid_extension * grid_length * block_height) >= height.
		 *
		 * That is,
		 *	grid_extension * grid_length >= ceil( height / block_height ).
		 *
		 * So, we can use:
		 *	grid_extension == grid_length == ceil( sqrt( ceil(height/block_height) ) ),
		 *
		 * but this may lead to an excessive number of idle thread blocks. That is,
		 *	(grid_extension * grid_length * block_height) >> height.
		 *
		 * Instead,
		 *	grid_extension	= ceil( height / (maxGridSizeX	 * block_height) ), which is << maxGridSizeY
		 *	grid_length	= ceil( height / (grid_extension * block_height) ), which is <= maxGridSizeX
		 *
		 * is more accurate.
		 *
		 * Note that grid_extension <= grid_length
		 */

		// Grid "extension"
		index_t const gh = maxGridSizeX * block_height;
		grid_extension = ( height + gh - 1 ) / gh;	// << maxGridSizeY

		// Grid "length"
		index_t const gw = grid_extension * block_height;
		grid_length = ( height + gw - 1 ) / gw;		// <= maxGridSizeX

		#if NMFGPU_PROFILING_KERNELS
			timing_index = 1;
		#endif

	} // If grid extension is required

	// ---------------------------

	#if NMFGPU_PROFILING_KERNELS
		start_cuda_timer();
	#endif

		idx_max( d_A, width, pitch, matrix_size, block_width, block_height, grid_extension, grid_length, stream_A, d_Idx );

			///////////////////////////////
			#if NMFGPU_DEBUG
				printf("\n--- [GPU%" PRI_IDX "] Resulting %s[i] = max(%s[i][..]) (height=%" PRI_IDX ",width=%" PRI_IDX
					", pitch=%" PRI_IDX ",block_width=%" PRI_IDX ",block_height=%" PRI_IDX ",grid_extension=%" PRI_IDX
					",grid_length=%" PRI_IDX ",transpose=%i): ---\n", device_id, matrix_name_Idx, matrix_name_A, height,
					width, pitch, block_width, block_height, grid_extension, grid_length, transpose );
				check_cuda_status();
				show_device_matrix( d_A, height, width, pitch, true, NULL );
			#endif
			///////////////////////////////

	#if NMFGPU_PROFILING_KERNELS
		stop_cuda_timer_cnt( &idx_max_timing[ timing_index ], matrix_size, 1 );
	#endif

} // matrix_idx_max

/////////////////////////////////////////////////////////////////////

/*
 * Transfers a matrix from the HOST (CPU) to the DEVICE (GPU) as a row vector.
 *
 * d_A[1..height][1..pitch] <--- A[1..height][1..pitch],
 *
 * If 'event_A' is non-NULL, the operation is recorded as an event.
 */
void upload_matrix( real const *__restrict__ A, index_t height, index_t pitch, real *__restrict__ d_A,
			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_VERBOSE_2
				index_t width, bool transpose, char const *__restrict__ const matrix_name_A,
				char const *__restrict__ const matrix_name_dA,
			#endif
			#if NMFGPU_PROFILING_TRANSF
				timing_data_t *__restrict__ const upload_timing,
			#endif
			hipStream_t stream_A, hipEvent_t *__restrict__ event_A )
{

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nUploading Matrix %s to %s (height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i, event %s)\n", matrix_name_A, matrix_name_dA, height, width, pitch, transpose,
				( event_A ? "provided" : "NOT provided"));
	#endif

	// ----------------------------------

	#if NMFGPU_SYNC_TRANSF
		// Synchronous data transfer: Waits until all previous operations have finished.
		check_cuda_status();
	#endif

	// ----------------------------------

	// Starts the transfer...

	size_t const nitems = height * pitch;

	#if NMFGPU_PROFILING_TRANSF
		start_cuda_timer();
	#endif
	{
		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
		hipError_t cuda_status =
		#endif

			hipMemcpyAsync( d_A, A, nitems * sizeof(real), hipMemcpyHostToDevice, stream_A );

			/* Same code using CUBLAS:
			 *	hipblasStatus_t cublas_status =
			 *		hipblasSetVectorAsync( nitems, sizeof(real), A, 1, d_A, 1, stream_A );
			 */

			///////////////////////////////
			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
				printf("\n--- [GPU%" PRI_IDX "] Uploaded matrix %s to %s (height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%"
					PRI_IDX ",transpose: %i, event %s): ---\n", device_id, matrix_name_A, matrix_name_dA, height, width,
					pitch, transpose, ( event_A ? "provided" : "NOT provided") );
				check_cuda_status_st( cuda_status );
				show_device_matrix( d_A, height, width, pitch, transpose, NULL );
			#elif NMFGPU_SYNC_TRANSF
				check_cuda_status_st( cuda_status );
			#endif
			/////////////////////////////
	}
	#if NMFGPU_PROFILING_TRANSF
		stop_cuda_timer_cnt( upload_timing, nitems, 1 );
	#endif

	// ----------------------------------

	// If 'event_A' is non-NULL, the operation is recorded as an event.
	if ( event_A ) {

		#if NMFGPU_DEBUG
			hipError_t cuda_status =
		#endif

			hipEventRecord( *event_A, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr, "\n[GPU%" PRI_IDX "] Error recording CUDA event: %s\nError in upload_matrix_event(%s to %s, "
						"height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX ",transpose: %i).\n", device_id,
						hipGetErrorString(cuda_status), matrix_name_A, matrix_name_dA, height, width, pitch,
						transpose );
			}
		#endif
		///////////////////////////////

	} // Records the operation as an event.

	// ----------------------------------

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nUploading Matrix %s to %s (height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i, event %s).. Done\n", matrix_name_A, matrix_name_dA, height, width, pitch,
				transpose, ( event_A ? "provided" : "NOT provided") );
	#endif

} // upload_matrix

/////////////////////////////////////////////////////////////////////

/*
 * Transfers (a portion of) a matrix from the HOST (CPU) to the DEVICE (GPU).
 *
 * d_A[1..height][1..block_pitch] <--- p_A[1..height][1..block_pitch],
 * where:
 *	p_A[1..height][1..block_pitch] == &A[X..(X+height)][offset..(offset+block_pitch)]
 *
 * block_pitch: Matrix block pitch.
 * block_width <= block_pitch
 * offset: Starting COLUMN.
 *
 * 0 <= offset < pitch.
 * Matrix is ROW-wise (i.e., it is NOT transposed).
 *
 * The transfer is delayed until the event 'event_A' has completed all previous operations.
 * Then, the operation is recorded using the same event object.
 *
 * It also checks that (offset + block_pitch) <= pitch,
 * and adjusts the width of the block to be transferred, if necessary.
 */
void upload_matrix_partial( real const *__restrict__ p_A, index_t height, index_t pitch, index_t offset,
				#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_VERBOSE_2
					index_t block_width, char const *__restrict__ const matrix_name_A,
					char const *__restrict__ const matrix_name_dA,
				#endif
				index_t block_pitch, real *__restrict__ d_A, hipStream_t stream_A, hipEvent_t event_A
				#if NMFGPU_PROFILING_TRANSF
					, timing_data_t *__restrict__ const upload_timing
				#endif
			)
{

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nUploading Matrix %s to %s (partial, height=%" PRI_IDX ",pitch=%" PRI_IDX ",offset=%" PRI_IDX ",block_width=%"
				PRI_IDX ", block_pitch=%" PRI_IDX ")\n", matrix_name_A, matrix_name_dA, height, pitch, offset, block_width,
				block_pitch);
	#endif


	#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
		hipError_t cuda_status = hipSuccess;
	#endif

	// ----------------------------------

	#if NMFGPU_SYNC_TRANSF
		// Synchronous data transfer: Waits until all previous operations have finished.
		check_cuda_status();
	#endif

	// ----------------------------------

	// Delays the transfer until the event has completed all previous operations.

	#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
		cuda_status =
	#endif

		hipStreamWaitEvent( stream_A, event_A, 0 );

		///////////////////////////////
		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr, "\n[GPU%" PRI_IDX "] Error setting CUDA event to wait for (hipStreamWaitEvent): %s\nError "
						"in upload_matrix_partial(%s to %s, partial, height=%" PRI_IDX ",pitch=%" PRI_IDX ",offset=%"
						PRI_IDX ",block_width=%" PRI_IDX ", block_pitch=%" PRI_IDX ").\n", device_id,
						hipGetErrorString(cuda_status), matrix_name_A, matrix_name_dA, height, pitch, offset,
						block_width, block_pitch);
			}
		#endif
		///////////////////////////////

	// ----------------------------------

	// Starts the transfer...

	#if NMFGPU_PROFILING_TRANSF
	start_cuda_timer();
	#endif

	if ( ( block_pitch < pitch ) + ( offset > 0 ) ) {

		/*
		 * It must be transferred as a 2D matrix.
		 */

		/* If necessary, adjusts the width to avoid an out-of-bound failure in CPU memory,
		 * but then, such width will NOT be a multiple of 'memory_alignment', resulting in a slower transfer.
		 */
		index_t const width = ( ( (offset + block_pitch) <= pitch ) ? block_pitch : (pitch - (offset + block_pitch)) );

		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
			cuda_status =
		#endif

			hipMemcpy2DAsync( d_A, block_pitch * sizeof(real), p_A, pitch * sizeof(real),
						width * sizeof(real), height, hipMemcpyHostToDevice, stream_A );

			/* Same code using CUBLAS:
			 *	hipblasStatus_t cublas_status =
			 *		hipblasSetMatrixAsync( width, height, sizeof(real), p_A, pitch, d_A, block_pitch, stream_A );
			 */


	// ( block_pitch == pitch ) && ( offset == 0 )
	} else {

		/*
		 * It can be transferred as a row vector.
		 */

		size_t const nitems = height * pitch;

		#if NMFGPU_PROFILING_TRANSF
		start_cuda_timer();
		#endif

			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
				cuda_status =
			#endif

				hipMemcpyAsync( d_A, p_A, nitems * sizeof(real), hipMemcpyHostToDevice, stream_A );

				/* Same code using CUBLAS:
				 *	hipblasStatus_t cublas_status =
				 *		hipblasSetVectorAsync( nitems, sizeof(real), p_A, 1, d_A, 1, stream_A );
				 */


	} // if ( ( block_pitch < pitch ) || ( offset > 0 ) )


		///////////////////////////////
		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
			printf("\n--- [GPU%" PRI_IDX "] Uploaded Matrix %s to %s (partial, height=%" PRI_IDX ",pitch=%" PRI_IDX ",offset=%"
				PRI_IDX ",block_width=%" PRI_IDX ", block_pitch=%" PRI_IDX ")\n", device_id,  matrix_name_A, matrix_name_dA,
				height, pitch, offset, block_width, block_pitch );
			check_cuda_status_st( cuda_status );
			show_device_matrix( d_A, height, block_width, block_pitch, false, NULL );
		#elif NMFGPU_SYNC_TRANSF
			check_cuda_status_st( cuda_status );
		#endif
		///////////////////////////////


	#if NMFGPU_PROFILING_TRANSF
	stop_cuda_timer_cnt( upload_timing, nitems, 1 );
	#endif


	// ----------------------------------

	// Records this operation using the same event object.

	#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
		cuda_status =
	#endif

		hipEventRecord( event_A, stream_A );

		///////////////////////////////
		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
			if ( cuda_status != hipSuccess ) {
				fflush(stdout);
				fprintf(stderr, "\n[GPU%" PRI_IDX "] Error recording CUDA: %s\nError in upload_matrix_partial(%s to %s, "
						"partial, height=%" PRI_IDX ",pitch=%" PRI_IDX ",offset=%" PRI_IDX ",block_width=%" PRI_IDX
						", block_pitch=%" PRI_IDX ").\n", device_id, hipGetErrorString(cuda_status), matrix_name_A,
						matrix_name_dA, height, pitch, offset, block_width, block_pitch);
			}
		#endif
		///////////////////////////////


	// ----------------------------------

	#if NMFGPU_VERBOSE_2
	if (! device_id)
		if ( ! device_id )
			printf("\nUploading Matrix %s to %s (partial, height=%" PRI_IDX ",pitch=%" PRI_IDX ",offset=%" PRI_IDX ",block_width=%"
				PRI_IDX ", block_pitch=%" PRI_IDX ")...Done.\n", matrix_name_A, matrix_name_dA, height, pitch, offset,
				block_width, block_pitch);
	#endif

} // upload_matrix_partial

/////////////////////////////////////////////////////////////////////

/*
 * Transfers a matrix from the DEVICE (GPU) to HOST (CPU), as a row vector.
 *
 * A[1..height][1..pitch] <--- d_A[1..height][1..pitch],
 */
void download_matrix( real *__restrict__ A, index_t height, index_t pitch, real const *__restrict__ d_A,
			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_VERBOSE_2
				index_t width, bool transpose, char const *__restrict__ const matrix_name_A,
				char const *__restrict__ const matrix_name_dA,
			#endif
			#if NMFGPU_PROFILING_TRANSF
				timing_data_t *__restrict__ const download_timing,
			#endif
			hipStream_t stream_A )
{

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nDownloading Matrix %s to %s (no event, height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i)\n", matrix_name_dA, matrix_name_A, height, width, pitch, transpose );
	#endif

	// ----------------------------------

	#if NMFGPU_SYNC_TRANSF
		// Synchronous data transfer: Waits until all previous operations have finished.
		check_cuda_status();
	#endif

	// ----------------------------------

	// Starts the transfer...

	size_t const nitems = height * pitch;

	#if NMFGPU_PROFILING_TRANSF
	start_cuda_timer();
	#endif

		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
		hipError_t cuda_status =
		#endif

			hipMemcpyAsync( A, d_A, nitems * sizeof(real), hipMemcpyDeviceToHost, stream_A );

			/* Same code using CUBLAS:
			 *	hipblasStatus_t cublas_status =
			 *		hipblasSetVectorAsync( nitems, sizeof(real), d_A, 1, A, 1, stream_A );
			 */

			///////////////////////////////
			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
				printf("\n--- [GPU%" PRI_IDX "] Downloaded matrix %s to %s (no event, height=%" PRI_IDX ",width=%"
					PRI_IDX ",pitch=%" PRI_IDX ",transpose: %i): ---\n", device_id, matrix_name_dA, matrix_name_A,
					height, width, pitch, transpose);
				check_cuda_status_st( cuda_status );
				if ( transpose )
					matrix_show( A, width, height, pitch, true, NULL );
				else
					matrix_show( A, height, width, pitch, false, NULL );
			#elif NMFGPU_SYNC_TRANSF
				check_cuda_status_st( cuda_status );
			#endif
			/////////////////////////////

	#if NMFGPU_PROFILING_TRANSF
	stop_cuda_timer_cnt( download_timing, nitems, 1 );
	#endif

	// ----------------------------------

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nDownloading Matrix %s to %s (no event, height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i)...Done.\n", matrix_name_dA, matrix_name_A, height, width, pitch, transpose );
	#endif

} // download_matrix

////////////////////////////////////////////////////////////////////////

/*
 * Transfers an INTEGER matrix from the DEVICE (GPU) to HOST (CPU), as a row vector.
 *
 * A[1..height][1..pitch] <--- d_A[1..height][1..pitch],
 */
void download_matrix_int( index_t *__restrict__ A, index_t height, index_t pitch, index_t const *__restrict__ d_A,
				#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_VERBOSE_2
					index_t width, bool transpose, char const *__restrict__ const matrix_name_A,
					char const *__restrict__ const matrix_name_dA,
				#endif
				#if NMFGPU_PROFILING_TRANSF
					timing_data_t *__restrict__ const download_timing,
				#endif
				hipStream_t stream_A )
{

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nDownloading integer Matrix %s to %s (no event, height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i)\n", matrix_name_dA, matrix_name_A, height, width, pitch, transpose );
	#endif

	// ----------------------------------

	#if NMFGPU_SYNC_TRANSF
		// Synchronous data transfer: Waits until all previous operations have finished.
		check_cuda_status();
	#endif

	// ----------------------------------

	// Starts the transfer...

	size_t const nitems = height * pitch;

	#if NMFGPU_PROFILING_TRANSF
	start_cuda_timer();
	#endif

		#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF || NMFGPU_SYNC_TRANSF
		hipError_t cuda_status =
		#endif

			hipMemcpyAsync( A, d_A, nitems * sizeof(index_t), hipMemcpyDeviceToHost, stream_A );

			/* Same code using CUBLAS:
			 *	hipblasStatus_t cublas_status =
			 *		hipblasSetVectorAsync( nitems, sizeof(real), d_A, 1, A, 1, stream_A );
			 */

			///////////////////////////////
			#if NMFGPU_DEBUG || NMFGPU_DEBUG_TRANSF
				printf("\n--- [GPU%" PRI_IDX "] Downloaded integer matrix %s to %s (no event, height=%" PRI_IDX ",width=%"
					PRI_IDX ",pitch=%" PRI_IDX ",transpose: %i): ---\n", device_id,  matrix_name_dA, matrix_name_A,
					height, width, pitch, transpose);
				check_cuda_status_st( cuda_status );
				if ( transpose )
					matrix_int_show( d_A, width, height, pitch, true, NULL );
				else
					matrix_int_show( d_A, height, width, pitch, false, NULL );
			#elif NMFGPU_SYNC_TRANSF
				check_cuda_status_st( cuda_status );
			#endif
			/////////////////////////////

	#if NMFGPU_PROFILING_TRANSF
	stop_cuda_timer_cnt( download_timing, nitems, 1 );
	#endif

	// ----------------------------------

	#if NMFGPU_VERBOSE_2
		if ( ! device_id )
			printf("\nDownloading integer Matrix %s to %s (no event, height=%" PRI_IDX ",width=%" PRI_IDX ",pitch=%" PRI_IDX
				",transpose: %i)...Done.\n", matrix_name_dA, matrix_name_A, height, width, pitch, transpose );
	#endif

} // download_matrix_int

////////////////////////////////////////////////////////////////////////
