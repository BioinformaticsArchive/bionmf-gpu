/************************************************************************
 * Copyright (C) 2011-2013:
 *
 *	Edgardo Mejia-Roa(*), Carlos Garcia, Jose Ignacio Gomez,
 *	Manuel Prieto, Francisco Tirado and Alberto Pascual-Montano(**).
 *
 *	(*)  ArTeCS Group, Complutense University of Madrid (UCM), Spain.
 *	(**) Functional Bioinformatics Group, Biocomputing Unit,
 *		National Center for Biotechnology-CSIC, Madrid, Spain.
 *
 *	E-mail for E. Mejia-Roa: <edgardomejia@fis.ucm.es>
 *	E-mail for A. Pascual-Montano: <pascual@cnb.csic.es>
 *
 *
 * This file is part of bioNMF-mGPU..
 *
 * BioNMF-mGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * BioNMF-mGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with BioNMF-mGPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 ***********************************************************************/
/**********************************************************
 * timing.cu
 *	Routines for timing and profiling.
 *
 * NOTE: The following macro constants can be defined to modify the
 *	behavior of routines, as well as some constant and data-type definitions.
 *
 *	Additional information:
 *		NMFGPU_VERBOSE_2: Shows the parameters in some routine calls.
 *
 *	Timing (WARNING: They PREVENT asynchronous operations):
 *		NMFGPU_PROFILING_TRANSF: Compute timing of data transfers (should be used with NMFGPU_SYNC_TRANSF).
 *		NMFGPU_PROFILING_KERNELS: Compute timing of CUDA kernels.
 *
 **********************************************************/

#include <stdlib.h>
#include <stdio.h>
#ifndef __STDC_FORMAT_MACROS
	#define __STDC_FORMAT_MACROS
#endif
#include <inttypes.h> /* PRIuMAX */

#include "real_type.h"
#include "timing.cuh"

// --------------------------------------
// --------------------------------------

#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS
	// CUDA Events for timing.
	hipEvent_t timing_events[ NUM_TIMING_EVENTS ];
#endif

#if NMFGPU_PROFILING_KERNELS
	// Timing on kernels
	timing_data_t reduce_timing[4], div_timing[2], mul_div_timing[2], adjust_timing[2], idx_max_timing[2], sub_timing[2];
#endif

#if NMFGPU_PROFILING_TRANSF
	// Timing on data transfers
	timing_data_t upload_Vrow_timing, upload_Vcol_timing, upload_H_timing, upload_W_timing, download_H_timing, download_W_timing,
			download_classf_timing;
#endif

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////

/*
 * Initializes kernel timers.
 */
void init_kernel_timers( void )
{

	#if NMFGPU_PROFILING_KERNELS

		for( index_t i=0 ; i<4 ; i++ ) {
			reduce_timing[i].time = 0.0;
			reduce_timing[i].counter = 0;
			reduce_timing[i].nitems = 0;
		}

		for( index_t i=0 ; i<2 ; i++ ) {
			div_timing[i].time = 0.0;
			div_timing[i].counter = 0;
			div_timing[i].nitems = 0;
		}

		for( index_t i=0 ; i<2 ; i++ ) {
			mul_div_timing[i].time = 0.0;
			mul_div_timing[i].counter = 0;
			mul_div_timing[i].nitems = 0;
		}

		for( index_t i=0 ; i<2 ; i++ ) {
			adjust_timing[i].time = 0.0;
			adjust_timing[i].counter = 0;
			adjust_timing[i].nitems = 0;
		}

		for( index_t i=0 ; i<2 ; i++ ) {
			idx_max_timing[i].time = 0.0;
			idx_max_timing[i].counter = 0;
			idx_max_timing[i].nitems = 0;
		}

		for( index_t i=0 ; i<2 ; i++ ) {
			sub_timing[i].time = 0.0;
			sub_timing[i].counter = 0;
			sub_timing[i].nitems = 0;
		}

	#endif	/* if defined( NMFGPU_PROFILING_KERNELS ) */

} // init_kernel_timers

/////////////////////////////////////////////////////////////////////

/*
 * Initializes timers for data-transfers.
 */
void init_transfer_timers( void )
{

	#if NMFGPU_PROFILING_TRANSF

		upload_Vrow_timing.time = 0.0;
		upload_Vrow_timing.counter = 0;
		upload_Vrow_timing.nitems = 0;

		upload_Vcol_timing.time = 0.0;
		upload_Vcol_timing.counter = 0;
		upload_Vcol_timing.nitems = 0;

		upload_H_timing.time = 0.0;
		upload_H_timing.counter = 0;
		upload_H_timing.nitems = 0;

		upload_W_timing.time = 0.0;
		upload_W_timing.counter = 0;
		upload_W_timing.nitems = 0;

		download_H_timing.time = 0.0;
		download_H_timing.counter = 0;
		download_H_timing.nitems = 0;

		download_W_timing.time = 0.0;
		download_W_timing.counter = 0;
		download_W_timing.nitems = 0;

		download_classf_timing.time = 0.0;
		download_classf_timing.counter = 0;
		download_classf_timing.nitems = 0;

	#endif	/* if defined( NMFGPU_PROFILING_TRANSF ) */

} // init_transfer_timers

/////////////////////////////////////////////////////////////////////

/*
 * Initializes the array of CUDA Events for timing.
 *
 * Returns EXIT_SUCCESS OR EXIT_FAILURE.
 */
int init_timing_events( void )
{

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		#if NMFGPU_VERBOSE_2
			printf("\n[GPU%" PRI_IDX "] Initializing array of CUDA Events for timing (number of events: %" PRI_IDX ")...\n",
				device_id, NUM_TIMING_EVENTS );
		#endif

		hipError_t cuda_status = hipSuccess;

		// ----------------------------

		// Start timer
		cuda_status = hipEventCreateWithFlags( &timing_events[ START_EVENT ], hipEventBlockingSync );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error creating CUDA event for timing (timer start): %s\n",
				device_id, hipGetErrorString(cuda_status) );
			return EXIT_FAILURE;
		}

		// Stop timer
		cuda_status = hipEventCreateWithFlags( &timing_events[ STOP_EVENT ], hipEventBlockingSync );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error creating CUDA event for timing (timer stop): %s\n",
				device_id, hipGetErrorString(cuda_status) );
			hipEventDestroy( timing_events[ START_EVENT ] );
			return EXIT_FAILURE;
		}

		// ----------------------------

		#if NMFGPU_VERBOSE_2
			printf("\n[GPU%" PRI_IDX "] Initializing array of CUDA Events for timing (number of events: %" PRI_IDX ")... Done.\n",
				device_id, NUM_TIMING_EVENTS );
		#endif

	#endif /* if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS */

	return EXIT_SUCCESS;

} // init_timing_events

/////////////////////////////////////////////////////////////////////

/*
 * Finalizes all CUDA Events for timing.
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE.
 */
int destroy_timing_events( void )
{

	int status = EXIT_SUCCESS;

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		#if NMFGPU_VERBOSE_2
			printf("\n[GPU%" PRI_IDX "] Finalizing CUDA Events for timing (%" PRI_IDX " objects)...\n",
				device_id, NUM_TIMING_EVENTS );
		#endif

		hipError_t cuda_status = hipSuccess;

		// ----------------------------

		// Stop timer
		cuda_status = hipEventDestroy( timing_events[ STOP_EVENT ] );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error destroying CUDA event for timing (timer stop): %s\n",
				device_id, hipGetErrorString(cuda_status) );
			status = EXIT_FAILURE;
		}

		// Start timer
		cuda_status = hipEventDestroy( timing_events[ START_EVENT ] );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error destroying CUDA event for timing (timer start): %s\n",
				device_id, hipGetErrorString(cuda_status) );
			status = EXIT_FAILURE;
		}

		// ----------------------------

		#if NMFGPU_VERBOSE_2
			printf("\n[GPU%" PRI_IDX "] Finalizing CUDA Events for timing (%" PRI_IDX " objects)... Done.\n",
				device_id, NUM_TIMING_EVENTS );
		#endif

	#endif /* if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS */

	return status;

} // destroy_timing_events

/////////////////////////////////////////////////////////////////////

/*
 * Starts the CUDA timer for the given CUDA event.
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE.
 */
int start_cuda_timer_ev( hipEvent_t timing_event )
{

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		hipError_t cuda_status = hipSuccess;

		cuda_status = hipDeviceSynchronize();
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] CUDA Error detected: %s\n", device_id, hipGetErrorString(cuda_status) );
			return EXIT_FAILURE;
		}

		cuda_status = hipEventRecord( timing_event, 0 );	// NULL stream. Waits for all operations.
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error recording a CUDA event: %s\n", device_id, hipGetErrorString(cuda_status) );
			return EXIT_FAILURE;
		}

	#endif	/* if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS */

	return EXIT_SUCCESS;

} // start_cuda_timer_ev

/////////////////////////////////////////////////////////////////////

/*
 * Starts the CUDA timer using the timing_events[ START_EVENT ] CUDA event.
 *
 * It is equivalent to: start_cuda_timer_ev( timing_events[ START_EVENT ] );
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE.
 */
int start_cuda_timer( void )
{

	int status = EXIT_SUCCESS;

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		status = start_cuda_timer_ev( timing_events[ START_EVENT ] );

	#endif

	return status;

} // start_cuda_timer

/////////////////////////////////////////////////////////////////////

/*
 * Stops the CUDA timer started using the given CUDA event.
 *
 * Returns the elapsed time (in ms) or a negative value on error.
 */
float stop_cuda_timer_ev( hipEvent_t start_timing_event )
{

	float elapsed_time = 0.0f;

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		hipError_t cuda_status = hipSuccess;

		hipEvent_t stop_timing_event = timing_events[ STOP_EVENT ];

		// ----------------------

		cuda_status = hipEventRecord( stop_timing_event, 0 );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] CUDA Error detected: %s\n", device_id, hipGetErrorString(cuda_status) );
			return -1.0f;
		}

		cuda_status = hipEventSynchronize( stop_timing_event );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] CUDA Error detected: %s\n", device_id, hipGetErrorString(cuda_status) );
			return -1.0f;
		}

		cuda_status = hipEventElapsedTime( &elapsed_time, start_timing_event, stop_timing_event );
		if ( cuda_status != hipSuccess ) {
			fflush(stdout);
			fprintf( stderr, "\n[GPU%" PRI_IDX "] Error retrieving elapsed time: %s\n", device_id, hipGetErrorString(cuda_status) );
			return -1.0f;
		}

	#endif	/* if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS */

	return elapsed_time;

} // stop_cuda_timer_ev

/////////////////////////////////////////////////////////////////////

/*
 * Stops the CUDA timer started using the timing_events[ START_EVENT ] CUDA event.
 *
 * It is equivalent to: stop_cuda_timer_ev( timing_events[ START_EVENT ] );
 *
 * Returns the elapsed time (in ms) or a negative value on error.
 */
float stop_cuda_timer( void )
{

	float elapsed_time = 0.0f;

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		elapsed_time = stop_cuda_timer_ev( timing_events[ START_EVENT ] );

	#endif

	return elapsed_time;

} // stop_cuda_timer

/////////////////////////////////////////////////////////////////////

/*
 * Stops the CUDA timer started using the given CUDA event.
 *
 * Updates the given timing data with the elapsed time (in ms).
 *
 * nitems: Number of items processed upon completion.
 * counter: Number of times that <nitems> items were processed.
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE.
 */
int stop_cuda_timer_cnt_ev( hipEvent_t start_timing_event, timing_data_t *__restrict__ td, index_t nitems, index_t counter )
{

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		float elapsed_time = stop_cuda_timer_ev( start_timing_event );
		if ( elapsed_time < 0.0f )
			return EXIT_FAILURE;

		if ( td ) {
			td->counter += (uintmax_t) counter;
			td->nitems  += (uintmax_t) nitems;
			td->time += (long double) elapsed_time;
		}

	#endif	/* if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS */

	return EXIT_SUCCESS;

} // stop_cuda_timer_cnt_ev

/////////////////////////////////////////////////////////////////////

/*
 * Stops the CUDA timer started using the timing_events[ START_EVENT ] CUDA event.
 *
 * Updates the given timing data with the elapsed time (in ms).
 *
 * nitems: Number of items processed upon completion.
 * counter: Number of times that <nitems> items were processed.
 *
 * Returns EXIT_SUCCESS or EXIT_FAILURE.
 */
int stop_cuda_timer_cnt( timing_data_t *__restrict__ td, index_t nitems, index_t counter )
{

	int status = EXIT_SUCCESS;

	#if NMFGPU_PROFILING_TRANSF || NMFGPU_PROFILING_KERNELS

		status = stop_cuda_timer_cnt_ev( timing_events[ START_EVENT ], td, nitems, counter );

	#endif

	return status;

} // stop_cuda_timer_cnt

/////////////////////////////////////////////////////////////////////

/*
 * Prints the following information for the given operation "<op>":
 *	- Total elapsed time, measured in milliseconds (but shown in seconds if show_secs is 'true').
 *	- Number of times the operation was performed and the average time, in milliseconds.
 *	- Bandwidth, in Gigabytes per second.
 *
 * size_of_data: Size, in bytes, of data processed.
 */
void print_elapsed_time( char const *__restrict__ const op, timing_data_t *__restrict__ td, size_t size_of_data, bool show_secs )
{

	// if ( op != NULL ) && ( td != NULL ) && ( size_of_data > 0 )
	if ( (size_t) op * (size_t) td * size_of_data ) {

		/* Bandwidth (GB/sec):
		 *	( (td->nitems * size_of_data) bytes / (2**30 bytes/GB) )  /  ( td->time (ms) / (1000 ms/sec) )
		 *
		 * Note that (size_of_data * ( 1000 / 2**30 )) is calculated at compile time.
		 */

		if ( show_secs )	// Time in seconds
			printf( "%s: %Lg sec", op, (td->time / 1000) );
		else
			printf( "%s: %Lg ms", op, td->time );

		printf( " (%" PRIuMAX " time(s), avg: %Lg ms), %Lg GiB/s\n", td->counter,
			(td->time / td->counter), ( ( td->nitems * size_of_data * (1000.0/(1<<30)) ) / td->time ) );
	}

} // print_elapsed_time

/////////////////////////////////////////////////////////////////////

/*
 * Shows time elapsed on some kernels.
 */
void show_kernel_times( void )
{

	#if NMFGPU_PROFILING_KERNELS

		printf("\n\tDevice Kernels:\n");

		bool const show_secs = false;	// Shows elapsed time in milliseconds, not in seconds.

		// --------------------

		// reduce (sum)
		{
			long double total_time = 0.0;
			index_t num_kernels = 0;
			if ( reduce_timing[0].counter ) {
				print_elapsed_time("\t\tGPU matrix_to_row", &reduce_timing[0], sizeof(real), show_secs );
				total_time = reduce_timing[0].time;
				num_kernels = 1;
			}
			if ( reduce_timing[1].counter ) {
				print_elapsed_time("\t\tGPU matrix_to_row (extended grid)", &reduce_timing[1], sizeof(real), show_secs );
				total_time += reduce_timing[1].time;
				num_kernels++;
			}
			if ( reduce_timing[2].counter ) {
				print_elapsed_time("\t\tGPU matrix_to_row (single block)", &reduce_timing[2], sizeof(real), show_secs );
				total_time += reduce_timing[2].time;
				num_kernels++;
			}
			if ( reduce_timing[3].counter ) {
				print_elapsed_time("\t\tGPU matrix_to_row (copy)", &reduce_timing[3], sizeof(real), show_secs );
				total_time += reduce_timing[3].time;
				num_kernels++;
			}
			if ( num_kernels > 1 )
				printf("\t\t\tTotal matrix_to_row time: %Lg ms\n", total_time );
		}

		// --------------------

		// div
		if ( div_timing[0].counter )
			print_elapsed_time("\t\tGPU div", &div_timing[0], sizeof(real), show_secs );

		if ( div_timing[1].counter )
			print_elapsed_time("\t\tGPU div (extended grid)", &div_timing[1], sizeof(real), show_secs );

		if ( div_timing[0].counter * div_timing[1].counter )
			printf("\t\t\tTotal div time: %Lg ms.\n", div_timing[0].time + div_timing[1].time );

		// ------------------

		// mul_div
		if ( mul_div_timing[0].counter )
			print_elapsed_time("\t\tGPU mul_div_time", &mul_div_timing[0], sizeof(real), show_secs );

		if ( mul_div_timing[1].counter )
			print_elapsed_time("\t\tGPU mul_div_time (extended grid)", &mul_div_timing[1], sizeof(real), show_secs );

		if ( mul_div_timing[0].counter * mul_div_timing[1].counter )
			printf("\t\t\tTotal mul_div time: %Lg ms.\n", mul_div_timing[0].time + mul_div_timing[1].time );


		// --------------------

		// Adjust
		if ( adjust_timing[0].counter )
			print_elapsed_time( "\t\tGPU adjust", &adjust_timing[0], sizeof(real), show_secs );

		if ( adjust_timing[1].counter )
			print_elapsed_time( "\t\tGPU adjust (extended grid)", &adjust_timing[1], sizeof(real), show_secs );

		if ( adjust_timing[0].counter * adjust_timing[1].counter )
			printf("\t\t\tTotal adjust time: %Lg ms.\n", adjust_timing[0].time + adjust_timing[1].time );

		// -------------------

		// Column index of maximum value.
		if ( idx_max_timing[0].counter )
			print_elapsed_time("\t\tGPU matrix_idx_max", &idx_max_timing[0], sizeof(index_t), show_secs );

		if ( idx_max_timing[1].counter )
			print_elapsed_time("\t\tGPU matrix_idx_max (extended grid)", &idx_max_timing[1], sizeof(index_t), show_secs );

		if ( idx_max_timing[0].counter * idx_max_timing[1].counter )
			printf("\t\t\tTotal matrix_idx_max time: %Lg ms.\n", idx_max_timing[0].time + idx_max_timing[1].time );

		// --------------------

		// sub
		if ( sub_timing[0].counter )
			print_elapsed_time("\t\tGPU sub", &sub_timing[0], sizeof(real), show_secs );

		if ( sub_timing[1].counter )
			print_elapsed_time("\t\tGPU sub (extended grid)", &sub_timing[1], sizeof(real), show_secs );

		if ( sub_timing[0].counter * sub_timing[1].counter )
			printf("\t\t\tTotal sub time: %Lg ms.\n", sub_timing[0].time + sub_timing[1].time );

	#endif	/* if defined( NMFGPU_PROFILING_KERNELS ) */

} // show_kernel_times

////////////////////////////////////////////////////////////////

/*
 * Shows time elapsed on data transfers.
 */
void show_transfer_times( void )
{

	#if NMFGPU_PROFILING_TRANSF

		printf("\n\tData Transfers:\n");

		bool const show_secs = true;	// Shows elapsed time in seconds.

		// --------------------

		print_elapsed_time( "\t\tSend V (rows)", &upload_Vrow_timing, sizeof(real), show_secs );

		if ( upload_Vcol_timing.counter )
			print_elapsed_time( "\t\tSend V (columns)", &upload_Vcol_timing, sizeof(real), show_secs );

		if ( upload_W_timing.counter )
			print_elapsed_time( "\t\tSend W", &upload_W_timing, sizeof(real), show_secs );

		if ( upload_H_timing.counter )
			print_elapsed_time( "\t\tSend H", &upload_H_timing, sizeof(real), show_secs );

		print_elapsed_time( "\t\tGet W", &download_W_timing, sizeof(real), show_secs );

		print_elapsed_time( "\t\tGet H", &download_H_timing, sizeof(real), show_secs );

		// Transfer of classification vector (test of convergence).
		print_elapsed_time( "\t\tGet Classification vector", &download_classf_timing, sizeof(index_t), show_secs );

		long double const total_data_transf = upload_Vrow_timing.time + upload_Vcol_timing.time +
							upload_W_timing.time + upload_H_timing.time +
							download_W_timing.time + download_H_timing.time +
							download_classf_timing.time;

		printf( "\tTotal data-transfers time: %Lg ms\n\n", total_data_transf );

	#endif /* defined( NMFGPU_PROFILING_TRANSF ) */

} // show_transfer_times

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
